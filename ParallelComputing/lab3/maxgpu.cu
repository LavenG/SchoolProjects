#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <locale.h>


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
__global__ void getmaxcu(unsigned int* num, unsigned int* max, int size){
  extern __shared__ unsigned int sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;

  //Load from global memory to shared memory
  sdata[tid] = num[i];
  if(i >= size){
    sdata[tid] = 0;
  }
  __syncthreads();
  // do reduction in shared memory by reducing each blocks max to a single value
  //strided index and non-divergent branch
  //reversed loop and threadID-based indexing
  for (int s=blockDim.x/2; s>0; s>>= 1) {
    __syncthreads();
    if (tid < s) {
      if(sdata[tid] < sdata[tid+s]){
        sdata[tid] = sdata[tid+s];
      }
    }

  }
  // write result for this block to global mem
  if(tid == 0){
    max[blockIdx.x] = sdata[0];
  }

}

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;

    //Get the propretiest of the specific device we're using
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    //Determine the maximum amount of threads in each block
    int maxThreadsPerBlock = devProp.maxThreadsPerBlock;

    //Determine the number of threads we need
    int numberOfThreads;
    if(size<maxThreadsPerBlock)
    {
      numberOfThreads = size;
    }else{
      numberOfThreads = maxThreadsPerBlock;
    }

    //Since we use the maximum amount of threads available per block,
    //in some cases, the last few threads of a block will be uselss
    //we ensure that this does not cause any problems by modyfying the
    //size accordingly and setting the indexes of the useless threads in the array
    //to 0
    int modSize = numberOfThreads;
    if(size % maxThreadsPerBlock != 0){
      modSize = (size/ maxThreadsPerBlock+1)* maxThreadsPerBlock;
    }else{
      modSize = size;
    }
    unsigned int *modNumbers = (unsigned int *)malloc(modSize* sizeof(unsigned int));
    for (int i = 0; i < modSize; i++) {
        if (i < size) {
            modNumbers[i] = numbers[i];
        } else {
            modNumbers[i] = 0;
        }
    }

    int numberOfBlocks = modSize/maxThreadsPerBlock;
    //array to send numbers to the device
    unsigned int * deviceArray;
    //Allocate the array on the device
    hipMalloc((void**)&deviceArray, modSize*(sizeof(unsigned int)));
    //Copy the values to the array on the device
    hipMemcpy(deviceArray, modNumbers, modSize*(sizeof(unsigned int)), hipMemcpyHostToDevice);
    //Array that stores the maximum of each block on the device
    unsigned int* deviceMax;
    //Allocate the array on the device
    hipMalloc((void**) &deviceMax, numberOfBlocks * sizeof(unsigned int));
    //Array that stores the maximum of each block on the host
    unsigned int* hostMax = (unsigned int *)malloc(numberOfBlocks*sizeof(unsigned int));

    //While we don't have a single value, keep calling the kernel so that we reduce
    //to a single block/value
    do {
      numberOfBlocks = ceil((float)modSize/(float)maxThreadsPerBlock);
      getmaxcu<<<numberOfBlocks, maxThreadsPerBlock, maxThreadsPerBlock*sizeof(unsigned int)>>>(deviceArray, deviceMax, modSize);
      //adjust the size since the array has been reduced
      modSize = numberOfBlocks;
      //set input = output from previous iteration
      deviceArray = deviceMax;
    } while(numberOfBlocks>1);

    //Transfer the max from device to host
    hipMemcpy(hostMax, deviceMax, numberOfBlocks*sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("The maximum number in the array is: %u\n", hostMax[0]);
    hipFree(deviceArray);
    hipFree(deviceMax);
    free(numbers);
    exit(0);
}
